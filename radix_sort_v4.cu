#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "Sorter.h"
#include <bitset>
#include <iostream>

#define BITS_LEN 32
#define BLOCK_SIZE 1024
#define MAX_LAYER 32
#define LAYER_SIZE 2 * BLOCK_SIZE // always multiple of BLOCK SIZE
// Bank Conflict
#define NUM_BANKS 32
#define LOG_NUM_BANKS 5
#define CONFLICT_FREE_OFFSET(n)((n) >> LOG_NUM_BANKS)

namespace v4 {

__global__ void preprocess_float(DEBUG_FLOAT* data) {
	int idx  = blockIdx.x * blockDim.x + threadIdx.x;
 	unsigned int data_temp = *(unsigned int *)(&data[idx]);    
    data_temp = (data_temp >> 31 & 0x1)? ~(data_temp): (data_temp) | 0x80000000;
	data[idx] = *(DEBUG_FLOAT *)&data_temp; 
}

__global__ void postprocess_float(DEBUG_FLOAT* const data) {
	int idx  = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int data_temp = *(unsigned int *)(&data[idx]);
    data_temp = (data_temp >> 31 & 0x1)? (data_temp) & 0x7fffffff: ~(data_temp);
	data[idx] = *(DEBUG_FLOAT *)&data_temp;
}

__global__ void reorder(
	DEBUG_FLOAT* data , 
	DEBUG_FLOAT* data_origin , 
	int n , 
	unsigned int* prefixSum_1_buffer,
	unsigned int* prefixSum_0_buffer,
	unsigned int* histgram_buffer,
	int num_layer,
	unsigned int *layer_offset,
	int sort_bit
	)
{
	int tid = threadIdx.x;
	int bid = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ DEBUG_FLOAT sdata[BLOCK_SIZE];
	__shared__ unsigned int slayer_offset[MAX_LAYER];
	sdata[tid] = data_origin[bid];
	if (tid < num_layer) slayer_offset[tid] = layer_offset[tid];
	__syncthreads();

	bool is_one = (__float_as_uint(sdata[tid]) >> sort_bit) & 1;
	int reordered_idx = histgram_buffer[1] * is_one;

	for (int layer = 0; layer < num_layer; ++layer) {
		if (is_one) reordered_idx += prefixSum_1_buffer[bid + slayer_offset[layer]];
		else reordered_idx += prefixSum_0_buffer[bid + slayer_offset[layer]];
		bid /= LAYER_SIZE;
	}					
	data[reordered_idx] = sdata[tid];
}


__global__ void init_sort(
	DEBUG_FLOAT* data_origin,
	unsigned int* prefixSum_1_buffer, 
	unsigned int* prefixSum_0_buffer,
	unsigned int sort_bit) 
{
	int tid = threadIdx.x;
	int bid  = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ DEBUG_FLOAT sdata[BLOCK_SIZE];
	sdata[tid] = data_origin[bid];

	bool is_one = (__float_as_uint(sdata[tid]) >> sort_bit) & 1;
	prefixSum_1_buffer[bid] = is_one;
	prefixSum_0_buffer[bid] = !is_one;
}


__global__ void block_prefix_sum(
	unsigned int* prefixSum_1_buffer, 
	unsigned int* prefixSum_0_buffer,
	unsigned int in_offset,
	unsigned int out_offset,
	unsigned int* histgram_buffer,
	bool isLast,
	unsigned int bit) 
{
	int tid = threadIdx.x;
	int bid = blockIdx.x * LAYER_SIZE;
	unsigned int last_1 = 0;
	unsigned int last_0 = 0;
	
	__shared__ unsigned int s1[LAYER_SIZE + CONFLICT_FREE_OFFSET(LAYER_SIZE)];
	__shared__ unsigned int s0[LAYER_SIZE + CONFLICT_FREE_OFFSET(LAYER_SIZE)];

	// Load Data
	int ai = tid, bi = tid + (LAYER_SIZE / 2);
	int bankOffsetA = CONFLICT_FREE_OFFSET(ai), bankOffsetB = CONFLICT_FREE_OFFSET(bi);

	s1[ai + bankOffsetA] = prefixSum_1_buffer[ai + bid + in_offset];
	s1[bi + bankOffsetB] = prefixSum_1_buffer[bi + bid + in_offset];

	s0[ai + bankOffsetA] = prefixSum_0_buffer[ai + bid + in_offset];
	s0[bi + bankOffsetB] = prefixSum_0_buffer[bi + bid + in_offset];

	if (tid == BLOCK_SIZE - 1) {
		last_1 = prefixSum_1_buffer[bi + bid + in_offset];
		last_0 = prefixSum_0_buffer[bi + bid + in_offset];
	}
	
	// Up-Sweep
	int offset = 1;
	for (int d = LAYER_SIZE >> 1; d > 0; d >>= 1) {
		__syncthreads();
		if (tid < d) {
			int ai = offset * (2 * tid + 1) - 1;
			int bi = offset * (2 * tid + 2) - 1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);
			s1[bi] += s1[ai];
			s0[bi] += s0[ai];
		}
		offset <<= 1;
	}

	// Down-Sweep
	if (tid == 0) {
		s1[LAYER_SIZE - 1 + CONFLICT_FREE_OFFSET(LAYER_SIZE - 1)] = 0;
		s0[LAYER_SIZE - 1 + CONFLICT_FREE_OFFSET(LAYER_SIZE - 1)] = 0;
	}
	
	for (int d = 1; d < LAYER_SIZE; d <<= 1) {
		offset >>= 1;
		__syncthreads();
		if (tid < d) {
			int ai = offset * (2 * tid + 1) - 1;
			int bi = offset * (2 * tid + 2) - 1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);

			unsigned int temp;
			temp = s1[ai];
			s1[ai] = s1[bi];
			s1[bi] += temp;
			
			temp = s0[ai];
			s0[ai] = s0[bi];
			s0[bi] += temp;
		}
	}
	__syncthreads();
	

	if (tid == BLOCK_SIZE - 1) {
		if (!isLast) {
			prefixSum_1_buffer[out_offset + blockIdx.x] = s1[LAYER_SIZE - 1 + CONFLICT_FREE_OFFSET(LAYER_SIZE - 1)] + last_1;
			prefixSum_0_buffer[out_offset + blockIdx.x] = s0[LAYER_SIZE - 1 + CONFLICT_FREE_OFFSET(LAYER_SIZE - 1)] + last_0;
		} else {
			histgram_buffer[1] = s0[LAYER_SIZE - 1 + CONFLICT_FREE_OFFSET(LAYER_SIZE - 1)] + last_0;
		}
	}

	prefixSum_1_buffer[ai + bid + in_offset] = s1[ai + bankOffsetA];
	prefixSum_1_buffer[bi + bid + in_offset] = s1[bi + bankOffsetB];
	prefixSum_0_buffer[ai + bid + in_offset] = s0[ai + bankOffsetA];
	prefixSum_0_buffer[bi + bid + in_offset] = s0[bi + bankOffsetB];
}

}

class RadixSorterv4 :public Sorter
{
	// Layerwise Prefix sum with Work-Efficient Sum Scan
	// Avoid Bank Conflict
public:
	
	void sort(DEBUG_FLOAT*& datas , int data_length) override{

		// Padding 
		int n = ceil(data_length, LAYER_SIZE) * LAYER_SIZE;
		int pad_len = n - data_length;

		// Init memory
		DEBUG_FLOAT *dev_data[2] = {0};		
		unsigned int *dev_histgram = 0;
		unsigned int *dev_layer_offset = 0;
		unsigned int *dev_1prefixSum = 0;
		unsigned int *dev_0prefixSum = 0;
		int pout = 0, pin = 1;

		// Buffer	    
		hipMalloc((void**)&dev_data[pout], n * sizeof(DEBUG_FLOAT));
		hipMemset(dev_data[pout] + data_length, 0xff,  pad_len * sizeof(DEBUG_FLOAT));
		hipMemcpy(dev_data[pout], datas, data_length * sizeof(DEBUG_FLOAT), hipMemcpyHostToDevice);

	    hipMalloc((void**)&dev_data[pin], n * sizeof(DEBUG_FLOAT));
		hipMemset(dev_data[pin] + data_length, 0xff,  pad_len * sizeof(DEBUG_FLOAT));
		hipMemcpy(dev_data[pin], datas, data_length * sizeof(DEBUG_FLOAT), hipMemcpyHostToDevice);

		// Histgram and prefix sum
		unsigned int prefix_sum_size = 0;
		unsigned int layer_offset[MAX_LAYER];
		int num_layer = 0;

		// Layer Initialization
		// n -> log_1024 (n) -> ... -> 1
		layer_offset[0] = 0;
		for (int layer_size = ceil(n, LAYER_SIZE); ; layer_size = ceil(layer_size, LAYER_SIZE)) {
			prefix_sum_size += layer_size * LAYER_SIZE;
			++num_layer;
			layer_offset[num_layer] = layer_offset[num_layer - 1] + layer_size * LAYER_SIZE;
			if (layer_size == 1) break;
		}
		prefix_sum_size++;


		hipMalloc((void**)&dev_histgram, 2 * sizeof(int));
		hipMalloc((void**)&dev_layer_offset, (num_layer + 1) * sizeof(int));
		hipMalloc((void**)&dev_1prefixSum, prefix_sum_size * sizeof(int));
		hipMalloc((void**)&dev_0prefixSum, prefix_sum_size * sizeof(int));
		
		// Initialize the buffer
		hipMemcpy(dev_layer_offset, layer_offset, (num_layer + 1) * sizeof(int), hipMemcpyHostToDevice);


		// Kernel launch
		int numBlocks = ceil(n, BLOCK_SIZE);

		// Prepare index and sortable conversion
		v4::preprocess_float<<< numBlocks , BLOCK_SIZE >>>(dev_data[pout]);
		
		for(int i = 0; i < BITS_LEN; ++i) {
			pout = 1 - pout; // swap double buffer indices
			pin = 1 - pout;

			hipMemset(dev_histgram , 0, 2 * sizeof(int));
			hipMemset(dev_1prefixSum, 0, prefix_sum_size * sizeof(int));
			hipMemset(dev_0prefixSum, 0, prefix_sum_size * sizeof(int));
			hipDeviceSynchronize();

			v4::init_sort<<< numBlocks , BLOCK_SIZE >>>(dev_data[pin],
														dev_1prefixSum,
														dev_0prefixSum,
														i);
			
			// Prefix_sum
			int num_prefix_sum_block = n;		
			
			for (int layer = 0; layer < num_layer; ++layer) {
				num_prefix_sum_block = ceil(num_prefix_sum_block, LAYER_SIZE);
				v4::block_prefix_sum<<< num_prefix_sum_block, BLOCK_SIZE >>>(dev_1prefixSum, 
																		dev_0prefixSum, 
																		layer_offset[layer], 
																		layer_offset[layer + 1], 
																		dev_histgram,
																		layer == num_layer - 1,
																		i);
			}
			// Sort
			v4::reorder<<< numBlocks , BLOCK_SIZE >>>(dev_data[pout],
													  dev_data[pin], 
													  n, 
													  dev_1prefixSum, 
													  dev_0prefixSum, 
													  dev_histgram,
													  num_layer,
													  dev_layer_offset, 
													  i);
			
		}
		
		
		// Copy origin_datas into sortable by index.
		v4::postprocess_float<<< numBlocks , BLOCK_SIZE >>>(dev_data[pout]);

		// Clear kernel
		hipMemcpy(datas, dev_data[pout] + pad_len, data_length * sizeof(float), hipMemcpyDeviceToHost);
		
		hipFree(dev_data[pout]);
		hipFree(dev_data[pin]);
		hipFree(dev_0prefixSum);
		hipFree(dev_1prefixSum);
		hipFree(dev_histgram);
		hipFree(dev_layer_offset);
		/*
		printf("============================= datas =================================\n");
		for(int i = 0 ; i < data_length ; i++){
			printBits(datas[i] );			
			printf(" : %f" , datas[i] );
			
    		printf("\n");
		}
		printf("\n");
		*/
	}
	
private:
	void printBits(DEBUG_FLOAT num) {
		  unsigned int bits = *reinterpret_cast<unsigned int*>(&num);

		// Print the bits
		std::bitset<32> bitset(bits);
		std::cout << "Bits of " << num << ": " << bitset << std::endl;
	}

	inline int ceil(int y, int b) { return (y + b - 1) / b; }
};
