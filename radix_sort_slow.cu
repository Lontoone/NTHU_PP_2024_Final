#include <stdio.h>
#include <stdlib.h>
#include "Sorter.h"

class SlowRadixSorter :public Sorter
{
public:
	
	void sort(float* datas , int data_length) override{
		int *dev_datas = 0;
	    hipMalloc((void**)&dev_datas, data_length * sizeof(float));
		hipMemcpy(dev_datas, datas, data_length * sizeof(float), hipMemcpyHostToDevice);
		// Init memory


		// Kernel launch
		// Preprocess float 
		// Post Process float

		// Clear kernel
		hipMemcpy(datas, dev_datas, data_length * sizeof(float), hipMemcpyHostToDevice);
		hipFree(dev_datas);
	}
private:
	
};
